#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include "text.h"
#include <iostream>

#define NB_ASCII_CHAR 128
const int threadsPerBlock = 256;


__global__
void histo_kernell( unsigned char *buffer , long size , unsigned int *histo ) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while (i < size) {
        atomicAdd(&(histo[buffer[i]]), 1);
        i += stride;
    }

}

__global__
void histo_kernell_private(unsigned char *buffer, long size, unsigned int *histo){
    __shared__ unsigned int histo_private[NB_ASCII_CHAR];

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    if (threadIdx.x < NB_ASCII_CHAR) histo_private[buffer[i]] = 0;
    __syncthreads();

    while (i < size) {
        atomicAdd(&(histo_private[buffer[i]]), 1);
        i += stride;
    }
    __syncthreads();

    if (i < NB_ASCII_CHAR)
        atomicAdd(&(histo[threadIdx.x]), histo_private[threadIdx.x]);
    __syncthreads();
}

int main( void ) {
    int len = strlen(h_str);
    printf("len:%d\n", len);
    int size = len*sizeof(char);
    const int blocksPerGrid = (size+threadsPerBlock-1) / threadsPerBlock ;

    // CPU computation
    u_int histo[NB_ASCII_CHAR] = {0};
    for (int i = 0; i < len; i++){
            histo[h_str[i]]++;
    }    
    //for (int bean = 0; bean < NB_ASCII_CHAR; bean++) {
    //    std::cout << (char) bean << " : " << histo[bean] << std::endl;
    //}

    // GPU computation
    u_int histo_gpu[NB_ASCII_CHAR] = {0};
    u_int *d_histo;
    u_char *d_str;
    hipMalloc( (void**)&d_histo, NB_ASCII_CHAR * sizeof(u_int) );
    hipMalloc( (void**)&d_str, size );
    hipMemcpy( d_str, h_str, size, hipMemcpyHostToDevice );
    hipMemcpy( d_histo, histo_gpu, NB_ASCII_CHAR * sizeof(u_int), hipMemcpyHostToDevice );
    histo_kernell_private<<<blocksPerGrid, threadsPerBlock>>>(d_str, size, d_histo);
    hipMemcpy( histo_gpu, d_histo, NB_ASCII_CHAR*sizeof(u_int), hipMemcpyDeviceToHost );
    for (int bean = 0; bean < NB_ASCII_CHAR; bean++) {
        std::cout << (char) bean << " : " << histo_gpu[bean] << " / " << histo[bean] << std::endl;
    }
    return 0;
}   
