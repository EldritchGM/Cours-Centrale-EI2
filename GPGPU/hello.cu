
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ const char *STR = "HELLO WORLD!";
const char STR_LENGTH = 12;

__global__ void hello()
{
	printf("%c", STR[threadIdx.x % STR_LENGTH]);
}

int main(void)
{
	int num_threads = STR_LENGTH;
	int num_blocks = 1;
	hello<<<num_blocks,num_threads>>>();
	hipDeviceSynchronize();
	printf("\n");
	return 0;
}
