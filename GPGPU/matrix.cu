#include "hip/hip_runtime.h"
 #include <iostream>
 #include <math.h>
 #include <assert.h>
 
 using namespace std;

 #define N (2048*2048)
 #define THREADS_PER_BLOCK 512

void initMatrix(float *m, int numRows, int numCols);
void computeMatrixMulCPU(float *A, float *B, float *C, int numCRows, int numCColumns, int itd_num);
void compareMatrix(float *A, float *B, int numRows, int numColumns);

int main(int argc, char *argv[])
{
    int numARows = atoi(argv[1]); // number of rows in the matrix A
    int numAColumns = atoi(argv[2]); // number of columns in the matrix A
    int numBRows = atoi(argv[3]); // number of rows in the matrix B
    int numBColumns = atoi(argv[4]); // number of columns in the matrix B
    int numCRows = numARows; // number of rows in the matrix C
    int numCColumns = numBColumns; // number of columns in the matrix C 
    assert(numAColumns == numBRows);

    float *A = (float *)malloc(numARows*numAColumns*sizeof(float));
    float *B = (float *)malloc(numBRows*numBColumns*sizeof(float));
    float *C = (float *)malloc(numCRows*numCColumns*sizeof(float));
    float *D = (float *)malloc(numCRows*numCColumns*sizeof(float));

    float *d_D;
    hipMalloc(void **) &d_D, numCRows*numCColumns*sizeof(float);

    // Initialize matrices on the host
    initMatrix(A, numARows, numAColumns);
    initMatrix(B, numBRows, numBColumns);

    computeMatrixMulCPU(A, B, C, numCRows, numCColumns, numAColumns);
    
    compareMatrix(C, C, numCRows, numCColumns);

    free(A);
    free(B);
    free(C);
    
    return 0;
}

void initMatrix(float *m, int numRows, int numCols){
    for (int i=0; i<numRows; i++){
        for (int j=0; j<numCols; j++){
            m[i*numCols+j] = sin(i*numCols+j);
        }
    }
}

void computeMatrixMulCPU(float *A, float *B, float *C, int numCRows, int numCColumns, int itd_num){
    // This function must return in C the result of the multiplication of the matrix A by the matrix B
    for (int i = 0; i < numCRows; i++){
        for (int j = 0; j < numCColumns; j++){
            for (int k = 0; k < itd_num; k++) {
                C[i * numCColumns + j] = A[i * itd_num + k] * B[k * numCColumns + j];
            }
        }
    }
}

__global__

void computeMatrixMulGPU(float *A, float *B, float *C, int numCRows, int numCColumns, int itd_num){
    int x = threadIdx.x + blockIdx.x * blockDim.y
    int y = threadIdx.x + blockIdx.y * blockDim.y

    if (x < numColumns && y < numCRows) {
        index_c = y * numCColumns + x
    }

    for (int k; k < itd_num; k++) {
        C[index_c] = A[y * numAColumns + k] * B[k * numCColumns + x]
    }
}

void compareMatrix(float *A, float *B, int numRows, int numColumns){
    for (int row = 0; row < numRows; row++){
        for (int col = 0; col < numColumns; col++){            
            assert(A[row*numColumns+col] == B[row*numColumns+col]);
        }
    }
    cout << "The matrices are identical" << endl;
}